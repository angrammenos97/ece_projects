#include "hip/hip_runtime.h"
#include "vptree.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

/////////////////////////////////
double* distance_from_last(double* X, int n, int dim)
{
	if (n == 1)
		exit(-1);
	double* d = (double*)malloc((n - 1) * sizeof(double));
	for (int i = 0; i < n - 1; i++) {
		*(d + i) = 0.0;
		for (int j = 0; j < dim; j++)
			*(d + i) += pow(*(X + i * dim + j) - *(X + (n - 1) * dim + j), 2);
		*(d + i) = sqrt(*(d + i));
	}
	return d;
}

void SWAP(double* X, double* d, int* idx, int dim, int a, int b)
{
	double tmpd;
	for (int j = 0; j < dim; j++) {
		tmpd = *(X + a * dim + j);
		*(X + a * dim + j) = *(X + b * dim + j);
		*(X + b * dim + j) = tmpd;
	}
	tmpd = *(d + a);
	*(d + a) = *(d + b);
	*(d + b) = tmpd;
	int tmpi = *(idx + a);
	*(idx + a) = *(idx + b);
	*(idx + b) = tmpi;
}

double quick_select(double* d, double* X, int* idx, int len, int k, int dim)
{
	int i, st;
	for (st = i = 0; i < len - 1; i++) {
		if (d[i] > d[len - 1]) continue;
		SWAP(X, d, idx, dim, i, st);
		st++;
	}
	SWAP(X, d, idx, dim, len - 1, st);
	return k == st ? d[st]
		: st > k ? quick_select(d, X, idx, st, k, dim)
		: quick_select(d + st, X + st * dim, idx + st, len - st, k - st, dim);
}

double median(double* X, int* idx, int n, int dim)
{
	if (n == 1)
		return 0.0;
	double* d = distance_from_last(X, n, dim);
	double md = quick_select(d, X, idx, (n - 1), (n - 2) / 2, dim);
	free(d);
	return md;
}

vptree* vpt(double* X, int* idx, int n, int dim)
{
	if (n == 0)
		return NULL;
	vptree* tree = (vptree*)malloc(sizeof(vptree));
	tree->vp = (X + (n - 1) * dim);
	tree->md = median(X, idx, n, dim);
	tree->idx = *(idx + n - 1);
	// split and recurse
	if ((n - 1) % 2 == 0) {
		tree->inner = vpt(X, idx, (n - 1) / 2, dim);
		tree->outer = vpt((X + ((n - 1) / 2) * dim), (idx + (n - 1) / 2), (n - 1) / 2, dim);
	}
	else {
		tree->inner = vpt(X, idx, (n - 1) / 2 + 1, dim);
		tree->outer = vpt((X + ((n - 1) / 2 + 1) * dim), (idx + (n - 1) / 2 + 1), (n - 1) / 2, dim);
	}
	return tree;
}
/////////////////////////////////

vptree* buildvp_cpu(double* X, int n, int d)
{
	double* X_copy = (double*)malloc(n * d * sizeof(double));
	int* idx = (int*)malloc(n * sizeof(int));
	for (int i = 0; i < n; i++) {
		*(idx + i) = i;
		for (int j = 0; j < d; j++)
			*(X_copy + i * d + j) = *(X + i * d + j);
	}

	clock_t start_t, end_t;
	start_t = clock();
	vptree* root = vpt(X_copy, idx, n, d);
	end_t = clock();
	printf("DONE in %lfmsec!\n", ((double)(end_t - start_t) / CLOCKS_PER_SEC) * 1000.0);

	return root;
}
