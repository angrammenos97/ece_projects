#include "hip/hip_runtime.h"
#include "device_global_memory.h"
#include <math.h>
#include <hip/hip_runtime.h>

//vptree.cu global variables
double* d_points;
double* d_pointsAux;
unsigned int* d_indexes;
unsigned int* d_indexesAux;
unsigned int* d_vpSwaps;
double* d_treeMDs;
unsigned int* d_treeIDXs;
unsigned int* d_nodesOffset;
unsigned int* d_nodesLength;

//distances.cu global variables
double* d_distances;

//quick_select.cu global variables
double* d_qsAux;
unsigned int* d_f;
unsigned int* d_t;
unsigned int* d_addr;
unsigned int* d_NFs;
char* d_e;

//knn_search.cu global variables
double* d_qpoints;
double* d_ndist;
unsigned int* d_nidx;
unsigned int* d_offsetsStack;
unsigned int* d_lengthsStack;
double* d_parentNDistStack;
double* d_parentMdStack;
char* d_isInnerStack;

/*Returns the smallest power of two*/
static unsigned int smallest_power_two(unsigned int n)
{
	unsigned int N = n;
	if ((N & (N - 1)) != 0) {	// fix if n is not power of 2
		N = 1;
		while (N < n)
			N <<= 1;
	}
	return N;
}

/*Functions to initialize memory*/
int qs_memory_allocate(unsigned int numberOfPoints, unsigned int maxParallelNodes)
{
	hipError_t err;
	unsigned int fixedNoP = smallest_power_two(numberOfPoints + 1);		//quick select needs length in powers of two
	//quick_select.cu global variables
	err = hipMalloc(&d_qsAux, fixedNoP * sizeof(double));										if (err != hipSuccess) return err;
	err = hipMalloc(&d_f, fixedNoP * sizeof(unsigned int));									if (err != hipSuccess) return err;
	err = hipMalloc(&d_t, fixedNoP * sizeof(unsigned int));									if (err != hipSuccess) return err;
	err = hipMalloc(&d_addr, fixedNoP * sizeof(unsigned int));									if (err != hipSuccess) return err;
	err = hipMalloc(&d_NFs, maxParallelNodes * sizeof(unsigned int));							if (err != hipSuccess) return err;
	err = hipMalloc(&d_e, fixedNoP * sizeof(char));											if (err != hipSuccess) return err;
	return hipSuccess;
}

int di_memory_allocate(unsigned int numberOfPoints)
{
	hipError_t err;
	unsigned int fixedNoP = smallest_power_two(numberOfPoints + 1);		//quick select needs length in powers of two
	//distances.cu global variables
	err = hipMalloc(&d_distances, fixedNoP * sizeof(double));									if (err != hipSuccess) return err;
	return hipSuccess;
}

int vp_memory_allocate(unsigned int numberOfPoints, unsigned int dimensionOfPoints)
{
	hipError_t err;
	unsigned int fixedNoP = smallest_power_two(numberOfPoints + 1);		//quick select needs length in powers of two
	unsigned int maxNodes = smallest_power_two(numberOfPoints + 1) / 2;	//max nodes on the last level of the tree
	//vptree.cu global variables
	err = hipMalloc(&d_points, (numberOfPoints * dimensionOfPoints) * sizeof(double));			if (err != hipSuccess) return err;
	err = hipMalloc(&d_pointsAux, (numberOfPoints * dimensionOfPoints) * sizeof(double));		if (err != hipSuccess) return err;
	err = hipMalloc(&d_indexes, numberOfPoints * sizeof(unsigned int));						if (err != hipSuccess) return err;
	err = hipMalloc(&d_indexesAux, numberOfPoints * sizeof(unsigned int));						if (err != hipSuccess) return err;
	err = hipMalloc(&d_vpSwaps, fixedNoP * sizeof(unsigned int));								if (err != hipSuccess) return err;
	err = hipMalloc(&d_treeMDs, numberOfPoints * sizeof(double));								if (err != hipSuccess) return err;
	err = hipMalloc(&d_treeIDXs, numberOfPoints * sizeof(unsigned int));						if (err != hipSuccess) return err;
	err = hipMalloc(&d_nodesOffset, maxNodes * sizeof(unsigned int));							if (err != hipSuccess) return err;
	err = hipMalloc(&d_nodesLength, maxNodes * sizeof(unsigned int));							if (err != hipSuccess) return err;
	return hipSuccess;
}

int knn_memory_allocate(unsigned int n, unsigned int m, unsigned int d, unsigned int k)
{
	hipError_t err;
	unsigned int maxDepth = (unsigned int)log2f(n) + 1;
	//knn_search.cu global variables
	err = hipMalloc(&d_points, (n * d) * sizeof(double));										if (err != hipSuccess) return err;
	err = hipMalloc(&d_qpoints, (m * d) * sizeof(double));										if (err != hipSuccess) return err;
	err = hipMalloc(&d_ndist, (m * k) * sizeof(double));										if (err != hipSuccess) return err;
	err = hipMalloc(&d_nidx, (m * k) * sizeof(unsigned int));									if (err != hipSuccess) return err;
	err = hipMalloc(&d_treeMDs, n * sizeof(double));											if (err != hipSuccess) return err;
	err = hipMalloc(&d_treeIDXs, n * sizeof(unsigned int));									if (err != hipSuccess) return err;
	err = hipMalloc(&d_offsetsStack, (m * maxDepth) * sizeof(unsigned int));					if (err != hipSuccess) return err;
	err = hipMalloc(&d_lengthsStack, (m * maxDepth) * sizeof(unsigned int));					if (err != hipSuccess) return err;
	err = hipMalloc(&d_parentNDistStack, (m * maxDepth) * sizeof(double));						if (err != hipSuccess) return err;
	err = hipMalloc(&d_parentMdStack, (m * maxDepth) * sizeof(double));						if (err != hipSuccess) return err;
	err = hipMalloc(&d_isInnerStack, (m * maxDepth) * sizeof(char));							if (err != hipSuccess) return err;
	return hipSuccess;
}

/*Functions to free memory*/
void qs_memory_deallocate()
{
	hipFree(d_qsAux);
	hipFree(d_f);
	hipFree(d_t);
	hipFree(d_addr);
	hipFree(d_e);
}

void di_memory_deallocate()
{
	hipFree(d_distances);
}

void vp_memory_deallocate()
{
	hipFree(d_points);
	hipFree(d_pointsAux);
	hipFree(d_indexes);
	hipFree(d_indexesAux);
	hipFree(d_vpSwaps);
	hipFree(d_treeMDs);
	hipFree(d_treeIDXs);
	hipFree(d_nodesOffset);
	hipFree(d_nodesLength);	
}

void knn_deallocate()
{
	hipFree(d_points);
	hipFree(d_qpoints);
	hipFree(d_ndist);
	hipFree(d_nidx);
	hipFree(d_treeMDs);
	hipFree(d_treeIDXs);
	hipFree(d_offsetsStack);
	hipFree(d_lengthsStack);
	hipFree(d_parentNDistStack);
	hipFree(d_parentMdStack);
	hipFree(d_isInnerStack);
}
