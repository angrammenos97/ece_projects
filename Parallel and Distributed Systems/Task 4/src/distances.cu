#include "hip/hip_runtime.h"
#include "distances.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

__device__ double* distances;
__device__ unsigned int numberOfColumns, numberOfRows, diMaxThreadsPerBlock;

__global__ void distance_kernel(double* points, unsigned int nodeOffset, unsigned int nodeLength)
{
	extern __shared__ double lastPoint[];
	unsigned int pntIdx = threadIdx.x;	//thread index
	//load last point into shared mem
	if (threadIdx.x < numberOfRows)
		for (unsigned int stride = 0; (stride + pntIdx) < numberOfRows; stride += blockDim.x)
			lastPoint[pntIdx + stride] = *(points + (nodeLength - 1) + ((pntIdx + stride) * numberOfColumns));
	__syncthreads();
	pntIdx = threadIdx.x + (blockIdx.x * blockDim.x);	//point index
	double pointDist, tempDiff;
	if (pntIdx < nodeLength - 1) {
		pointDist = 0.0;
		for (unsigned int d = 0; d < numberOfRows; d++) {
			tempDiff = *(points + pntIdx + d * numberOfColumns) - lastPoint[d];
			pointDist += tempDiff * tempDiff;
		}
		distances[nodeOffset + pntIdx] = sqrt(pointDist);		//save result back to global mem
	}
}

__device__ void distance_from_last(double* points, unsigned int nodeOffset, unsigned int nodeLength, hipStream_t nodeStream)
{
	unsigned int totalThreads = (numberOfRows > (nodeLength - 1)) ? numberOfRows : nodeLength - 1;
	unsigned int blockSz = (totalThreads < diMaxThreadsPerBlock) ? totalThreads : diMaxThreadsPerBlock;
	unsigned int gridSz = (totalThreads + blockSz - 1) / blockSz;
	distance_kernel <<<gridSz, blockSz, numberOfRows * sizeof(double), nodeStream>>> (points, nodeOffset, nodeLength);	//"+numberOfRows" to hold the last point in shared mem
}

__global__ void distance_init_kernel(double* d_distances, unsigned int numberOfPoints, unsigned int dimensionOfPoints, unsigned int maxThreadsPerBlock)
{
	if ((threadIdx.x) == 0 && (blockIdx.x == 0)) {
		numberOfColumns = numberOfPoints;		numberOfRows = dimensionOfPoints;		diMaxThreadsPerBlock = maxThreadsPerBlock;
		//Initialize device pointers to global memory
		distances = d_distances;		
	}
}
