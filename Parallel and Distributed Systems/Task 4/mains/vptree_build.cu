#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "sys/time.h"
#include "vptree.h"

#define DefaultNumPoints 1000
#define DefaultDim 20

double* X;
unsigned int nop = DefaultNumPoints;
unsigned int dim = DefaultDim;
int matlab = 0;

struct timeval startwtime, endwtime;

void help(int argc, char* argv[]);
void export_data(FILE* file);
void export_struct(FILE* file, vptree* root, const char* root_name, unsigned int str_size);

int main(int argc, char* argv[])
{
	help(argc, argv);
	printf("Running with values n=%i and d=%i\n", nop, dim);

	srand((unsigned int)32);//time(NULL));
	FILE* data = NULL;

	// Generate random point set
	printf("Generating random data set... ");
	gettimeofday(&startwtime, NULL);
	X = (double*)malloc(nop * dim * sizeof(double));
	for (unsigned int i = 0; i < nop * dim; i++)
		*(X + i) = ((double)rand() / (RAND_MAX));

	gettimeofday(&endwtime, NULL);
	double p_time = (double)((endwtime.tv_usec - startwtime.tv_usec) / 1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
	printf("DONE in %fsec!\n", p_time);

	if (matlab) {
		printf("Writting dataset to data.m... ");
		gettimeofday(&startwtime, NULL);
		data = fopen("./data.m", "w");
		export_data(data);
		fclose(data);
		gettimeofday(&endwtime, NULL);
		p_time = (double)((endwtime.tv_usec - startwtime.tv_usec) / 1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
		printf("DONE in %fsec!\n", p_time);
	}

	// Build search tree
	printf("Building search tree... ");
	gettimeofday(&startwtime, NULL);
	vptree* tree = buildvp(X, nop, dim);
	gettimeofday(&endwtime, NULL);
	p_time = (double)((endwtime.tv_usec - startwtime.tv_usec) / 1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
	printf("DONE in %fsec!\n", p_time);

	if (matlab) {
		printf("Appending tree to data.m... ");
		gettimeofday(&startwtime, NULL);
		data = fopen("./data.m", "a");
		export_struct(data, tree, "tree", 5);
		fclose(data);
		gettimeofday(&endwtime, NULL);
		p_time = (double)((endwtime.tv_usec - startwtime.tv_usec) / 1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
		printf("DONE in %fsec!\n", p_time);
	}

	printf("Exiting\n");
	free(X);
	return 0;
}


void help(int argc, char* argv[])
{
	if (argc > 1) {
		for (int i = 1; i < argc; i += 2) {
			if (*argv[i] == '-') {
				if (*(argv[i] + 1) == 'n')
					nop = atoi(argv[i + 1]);
				else if (*(argv[i] + 1) == 'd')
					dim = atoi(argv[i + 1]);
				else if (*(argv[i] + 1) == 'm') {
					matlab = 1;
					i--;
				}
				else {
					help(1, argv);
					return;
				}
			}
			else {
				help(1, argv);
				return;
			}
		}
		return;
	}
	printf("Flags to use:\n");
	printf("-n [Number] :Number of points (default:%i)\n", DefaultNumPoints);
	printf("-d [Dimension] :Dimension of the space (default: %i)\n", DefaultDim);
	printf("-m :Print results into data.m file to evaluate in MATLAB\n");
}

void export_data(FILE* file)
{
	fprintf(file, "n = %i;\n", nop);
	fprintf(file, "dim = %i;\n", dim);
	fprintf(file, "X=[");
	for (unsigned int i = 0; i < nop; i++) {
		fprintf(file, "[");
		for (unsigned int j = 0; j < dim; j++)
			fprintf(file, "%lf ", *(X + (i * dim) + j));
		fprintf(file, "]; ");
	}
	fprintf(file, "];\n");
}

void export_struct(FILE* file, vptree* root, const char* root_name, unsigned int str_size)
{
	if (root == NULL) {
		fprintf(file, "%s = [];\n", root_name);
		return;
	}
	else {
		fprintf(file, "%s.vp = [", root_name);
		for (unsigned int j = 0; j < dim; j++)
			fprintf(file, "%lf ", *(X + (getIDX(root) * dim) + j));
		fprintf(file, "];\n");
		fprintf(file, "%s.md = %lf;\n", root_name, getMD(root));
		fprintf(file, "%s.idx = %i;\n", root_name, getIDX(root) + 1);
		char* tmpi = (char*)malloc((str_size + 6) * sizeof(char));
		memcpy(tmpi, root_name, str_size - 1);
		memcpy(tmpi + str_size - 1, ".inner", 7);
		char* tmpo = (char*)malloc((str_size + 6) * sizeof(char));
		memcpy(tmpo, root_name, str_size - 1);
		memcpy(tmpo + str_size - 1, ".outer", 7);
		export_struct(file, getInner(root), tmpi, str_size + 6);
		export_struct(file, getOuter(root), tmpo, str_size + 6);
	}
}
